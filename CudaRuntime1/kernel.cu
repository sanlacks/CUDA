#include "hip/hip_runtime.h"
﻿#include <math.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <fstream>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <complex>
#include <iostream>
#include <hip/hip_runtime.h>
#include "matplotlibcpp.h" 
#include ""

using namespace std;
namespace plt = matplotlibcpp;


//读取CSV文件并返回行数
void readCSV(std::vector<float>& hostData, std::ifstream& file, int& lineCount) {
    std::string line;
    lineCount = 0;

    while (std::getline(file, line)) {
        lineCount++;
        hostData.push_back(std::stof(line));
    }
}

// 滑动平均滤波函数
std::vector<float> movingAverage(const std::vector<float>& data, int windowSize) {
    std::vector<float> smoothedData;
    int dataLength = data.size();

    for (int i = 0; i < dataLength; ++i) {
        float sum = 0.0f;
        int count = 0;

        for (int j = i - windowSize / 2; j <= i + windowSize / 2; ++j) {
            if (j >= 0 && j < dataLength) {
                sum += data[j];
                count++;
            }
        }

        float average = sum / count;
        smoothedData.push_back(average);
    }

    return smoothedData;
}

int main()
{
    // 打开CSV文件
    std::ifstream file("D:\\Codes\\VisualStudio\\signal13.csv");

    // 检查文件是否成功打开
    if (!file.is_open()) {
        std::cerr << "无法打开文件" << std::endl;
        return 1;
    }

    int LENGTH = 0;
    vector<float> Data(LENGTH);

    //读取csv文件
    readCSV(Data, file, LENGTH);
    // 对原始数据应用滑动平均滤波
    int windowSize1 = 10; // 设置滑动窗口大小
    std::vector<float> smoothedData = movingAverage(Data, windowSize1);
    

    // 分配和传输数据到CUDA设备
    hipfftComplex* CompData = (hipfftComplex*)malloc(LENGTH * sizeof(hipfftComplex));//allocate memory for the data in host
 
    for (int i = 0; i < LENGTH; i++)
    {
        CompData[i].x = Data[i];
        CompData[i].y = 0;
    }

    hipfftComplex* d_fftData;
    hipMalloc((void**)&d_fftData, LENGTH * sizeof(hipfftComplex));// allocate memory for the data in device
    hipMemcpy(d_fftData, CompData, LENGTH * sizeof(hipfftComplex), hipMemcpyHostToDevice);// copy data from host to device

    // 创建cuFFT计划
    hipfftHandle plan;// cuda library function handle
    hipfftPlan1d(&plan, LENGTH, HIPFFT_C2C, 1);//declaration

    // 执行FFT
    hipfftExecC2C(plan, (hipfftComplex*)d_fftData, (hipfftComplex*)d_fftData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done

    // 传输FFT结果到主机
    hipMemcpy(CompData, d_fftData, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);// copy the result from device to host


    //double fs = 1 / 0.0001220703125; //采样率
    double fs = 100000; //采样率

    // 创建频率轴
    int n = LENGTH / 2;
    std::vector<float> x(n), y(n);
    for (int i = 0; i < LENGTH / 2; ++i)
    {
        x.at(i) = fs * i / LENGTH;
        y.at(i) = abs(CompData[i].x) * 2.0 / LENGTH;
    }

    // 滑动平均滤波
    int windowSize2 = 5; // 设置滑动窗口大小
    std::vector<float> smoothedY = movingAverage(y, windowSize2);

    // 使用matplotlibcpp绘制结果
    plt::plot(x, smoothedY); // 使用滤波后的数据进行绘图
    plt::xlim(-100, 3000);
    plt::xlabel("Frequency/hz");
    plt::ylabel("Amplitude");
    plt::grid('b');

    plt::show();
    hipfftDestroy(plan);
    free(CompData);
    hipFree(d_fftData);

    return 0;
}