#include "hip/hip_runtime.h"
﻿#include <hipfft/hipfft.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <complex>
#include <hip/hip_runtime.h>
#include "matplotlibcpp.h" 

#include "movingAverage.hpp"
#include "readCSV.hpp"
#include "lm.hpp"


using namespace Eigen;
using namespace std;
namespace plt = matplotlibcpp;



int main()
{
    // 打开CSV文件
    ifstream file("D:\\Codes\\VisualStudio\\data.csv");

    // 检查文件是否成功打开
    if (!file.is_open()) {
        cerr << "无法打开文件" << endl;
        return 1;
    }

    int LENGTH = 0;
    
    vector<double> x_data(LENGTH);
    vector<double> y_data(LENGTH);
    vector<double> lmy_data(LENGTH);

    //读取csv文件
    readCSV(x_data, y_data, file, LENGTH);

    // 对原始数据应用滑动平均滤波
    int windowSize1 = 5; // 设置滑动窗口大小

    int dataLength = y_data.size();

    vector<double> smoothedy_data = movingAverage(y_data, windowSize1);

    //writeCSV(smoothedy_data, "sy_data.csv");

    // lm参数计算
    Vector4d para = fit_curve(x_data, smoothedy_data);
    cout << "The optimal parameters are: " << para.transpose() << endl;

    //lm拟合
    int i = 0;
    int Nl = x_data.size();

    for (i=0;i<Nl;i++){
        
        lmy_data.push_back(func(x_data[i], para));

    }

    //writeCSV(lmy_data, "fitsy_data.csv");
    

    //plt::named_plot("Origin Data", x_data, y_data,"y");// 原始数据绘图
    
    //plt::named_plot("Smooth Data", x_data, smoothedy_data, "b");// 滤波数据绘图
    
    //plt::named_plot("LM Data", x_data, lmy_data, "r");// LM数据绘图
   
    //plt::title("Data Process");
    //plt::legend();
    //plt::show();
   
    

    

    // 将 lmy_data 复制到 GPU 内存
    double* d_lmy_data;
    hipMalloc((void**)&d_lmy_data, Nl * sizeof(double));
    hipMemcpy(d_lmy_data, lmy_data.data(), Nl * sizeof(double), hipMemcpyHostToDevice);
    
    // 创建 CUFFT 计划
    hipfftHandle plan;
    hipfftPlan1d(&plan, Nl, HIPFFT_Z2Z, 1);

    // 执行 FFT
    hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_lmy_data, (hipfftDoubleComplex*)d_lmy_data, HIPFFT_FORWARD);

    // 将结果传回主机
    hipMemcpy(lmy_data.data(), d_lmy_data, Nl * sizeof(double), hipMemcpyDeviceToHost);

    // 销毁 CUFFT 计划和 GPU 内存
    hipfftDestroy(plan);
    hipFree(d_lmy_data);

    //double fs = 1 / 0.0001220703125; //采样率
    double fs = 1000; //采样率

    
    // 创建频率轴
    int n = Nl/2;
    std::vector<double> x(n), y(n);
    
    for (int i = 0; i < Nl/2; ++i){

        x.at(i) = fs * i / Nl;
        y.at(i) = abs(lmy_data[i]) * 2.0 / Nl;

    }
    // 使用max_element 查找最大值的迭代器
    auto maxElement = max_element(y.begin(), y.end());
   
    cout << "最大值是: " << *maxElement << std::endl;
   
    

    // 滑动平均滤波
    int windowSize2 = 5; // 设置滑动窗口大小
    std::vector<double> smoothedY = movingAverage(y, windowSize2);

    // 使用matplotlibcpp绘制结果   
    plt::plot(x, smoothedY); // 使用滤波后的数据进行绘图
    //plt::xlim(-100, 3000);
    //plt::xlabel("Frequency/hz");
    //plt::ylabel("Amplitude");
   // plt::grid('b');
    plt::show();
    

    return 0;
}