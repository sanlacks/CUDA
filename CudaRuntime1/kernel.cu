#include "hip/hip_runtime.h"
﻿#include <math.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <fstream>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <complex>
#include <iostream>
#include <hip/hip_runtime.h>
#include "matplotlibcpp.h" 
#include ""

using namespace std;
namespace plt = matplotlibcpp;


//读取CSV文件并返回行数
void readCSV(std::vector<float>& hostData, std::ifstream& file, int& lineCount) {
    std::string line;
    lineCount = 0;

    while (std::getline(file, line)) {
        lineCount++;
        hostData.push_back(std::stof(line));
    }
}


int main()
{
    // 打开CSV文件
    std::ifstream file("D:\\Codes\\VisualStudio\\signal13.csv");

    // 检查文件是否成功打开
    if (!file.is_open()) {
        std::cerr << "无法打开文件" << std::endl;
        return 1;
    }

    int LENGTH = 0;
    vector<float> Data(LENGTH);

    //读取csv文件
    readCSV(Data, file, LENGTH);

    

    // 分配和传输数据到CUDA设备
    hipfftComplex* CompData = (hipfftComplex*)malloc(LENGTH * sizeof(hipfftComplex));//allocate memory for the data in host
 
    for (int i = 0; i < LENGTH; i++)
    {
        CompData[i].x = Data[i];
        CompData[i].y = 0;
    }

    hipfftComplex* d_fftData;
    hipMalloc((void**)&d_fftData, LENGTH * sizeof(hipfftComplex));// allocate memory for the data in device
    hipMemcpy(d_fftData, CompData, LENGTH * sizeof(hipfftComplex), hipMemcpyHostToDevice);// copy data from host to device

    // 创建cuFFT计划
    hipfftHandle plan;// cuda library function handle
    hipfftPlan1d(&plan, LENGTH, HIPFFT_C2C, 1);//declaration

    // 执行FFT
    hipfftExecC2C(plan, (hipfftComplex*)d_fftData, (hipfftComplex*)d_fftData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done

    // 传输FFT结果到主机
    hipMemcpy(CompData, d_fftData, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);// copy the result from device to host


    //double fs = 1 / 0.0001220703125; //采样率
    double fs = 100000; //采样率
    //for (i = 0; i < LENGTH / 2; i++)
    //{
    //    printf("i=%d\tf= %6.1fHz\tRealAmp=%3.1f\t", i, fs * i / LENGTH, CompData[i].x * 2.0 / LENGTH);
    //    printf("ImagAmp=+%3.1fi", CompData[i].y * 2.0 / LENGTH);
    //    printf("\n");
    //}

    //创建频率轴
    int n = LENGTH / 2;
    std::vector<double> x(n), y(n);
    for (int i = 0; i < LENGTH / 2; ++i) {
        x.at(i) = fs * i / LENGTH;
        y.at(i) = abs(CompData[i].x) * 2.0 / LENGTH;

    }

    //在C++中使用matplotlibcpp绘制FFT结果的振幅谱
    plt::plot(x, y);
    //plt::pause(2);
    plt::xlim(-10, 10000);
    plt::xlabel("Frequency/hz");
    plt::ylabel("Amplitude");
    plt::grid('b');

    plt::show();
    hipfftDestroy(plan);
    free(CompData);
    hipFree(d_fftData);

    return 0;
}