#include "hip/hip_runtime.h"
﻿#include <hipfft/hipfft.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <complex>
#include <hip/hip_runtime.h>
#include "matplotlibcpp.h" 
#include "fftw3.h"
#include "movingAverage.hpp"
#include "readCSV.hpp"
#include "lm.hpp"


using namespace Eigen;
using namespace std;
namespace plt = matplotlibcpp;



int main()
{
    // 打开CSV文件
    ifstream file1("D:\\Codes\\VisualStudio\\datas\\data1.csv");
    ifstream file2("D:\\Codes\\VisualStudio\\datas\\data2.csv");
    ifstream file3("D:\\Codes\\VisualStudio\\datas\\data3.csv");
    ifstream file4("D:\\Codes\\VisualStudio\\datas\\data4.csv");


    // 检查文件是否成功打开
    //if (!file.is_open()) {
    //    cerr << "无法打开文件" << endl;
    //    return 1;
    //}

    int LENGTH = 0;
    
    vector<double> x_data1(LENGTH);
    vector<double> y_data1(LENGTH);
    vector<double> lmy_data1(LENGTH);
    

    vector<double> x_data2(LENGTH);
    vector<double> y_data2(LENGTH);
    vector<double> lmy_data2(LENGTH);

    vector<double> x_data3(LENGTH);
    vector<double> y_data3(LENGTH);
    vector<double> lmy_data3(LENGTH);

    vector<double> x_data4(LENGTH);
    vector<double> y_data4(LENGTH);
    vector<double> lmy_data4(LENGTH);

    //读取csv文件
    readCSV(x_data1, y_data1, file1, LENGTH);

    readCSV(x_data2, y_data2, file2, LENGTH);

    readCSV(x_data3, y_data3, file3, LENGTH);

    readCSV(x_data4, y_data4, file4, LENGTH);

    //plt::named_plot("Data 1", x_data1, y_data1);// Data1数据绘图

    //plt::named_plot("Data 2", x_data2, y_data2);// Data2数据绘图

    //plt::named_plot("Data 3", x_data3, y_data3);// Data3数据绘图

    //plt::named_plot("Data 3", x_data4, y_data4);// Data4数据绘图
    //plt::title("Data Signal");
   
    //plt::show();

    // 对原始数据应用滑动平均滤波
    int windowSize1 = 5; // 设置滑动窗口大小

    int n = y_data1.size();

    vector<double> smoothedy_data1 = movingAverage(y_data1, windowSize1);

    vector<double> smoothedy_data2 = movingAverage(y_data2, windowSize1);

    vector<double> smoothedy_data3 = movingAverage(y_data3, windowSize1);

    vector<double> smoothedy_data4 = movingAverage(y_data4, windowSize1);

    //writeCSV(smoothedy_data, "sy_data.csv");

    // lm参数计算
    Vector4d para1 = fit_curve(x_data1, smoothedy_data1);
    //cout << "Data 1's parameters are: " << para1.transpose() << endl;
    Vector4d para2 = fit_curve(x_data2, smoothedy_data2);
    //cout << "Data 2's parameters are: " << para2.transpose() << endl;
    Vector4d para3 = fit_curve(x_data3, smoothedy_data3);
    //cout << "Data 3's parameters are: " << para3.transpose() << endl;
    Vector4d para4 = fit_curve(x_data4, smoothedy_data4);
    //cout << "Data 4's parameters are: " << para4.transpose() << endl;

    //lm拟合
    int i = 0;
    
    for (i = 0; i < n; i++){   
        lmy_data1.push_back(func(x_data1[i], para1));
    }

    for (i = 0; i < n; i++) {
        lmy_data2.push_back(func(x_data2[i], para2));
    }

    for (i = 0; i < n; i++) {
        lmy_data3.push_back(func(x_data3[i], para3));
    }

    for (i = 0; i < n; i++) {
        lmy_data4.push_back(func(x_data4[i], para4));
    }

    //writeCSV(lmy_data, "fitsy_data.csv");
    

    //plt::named_plot("Origin Data", x_data1, y_data1,":");// 原始数据绘图
    //plt::named_plot("Smooth Data", x_data1, smoothedy_data1, "--");// 滤波数据绘图
    //plt::named_plot("LM Data", x_data1, lmy_data1, "r");// LM数据绘图
    //plt::title("Data 1 Process");
    //plt::legend();
    //plt::show();
   
    //plt::named_plot("Origin Data", x_data2, y_data2, ":");// 原始数据绘图
    //plt::named_plot("Smooth Data", x_data2, smoothedy_data2, "--");// 滤波数据绘图
    //plt::named_plot("LM Data", x_data2, lmy_data2, "r");// LM数据绘图
    //plt::title("Data 2 Process");
    //plt::legend();
    //plt::show();

    //plt::named_plot("Origin Data", x_data3, y_data3, ":");// 原始数据绘图
    //plt::named_plot("Smooth Data", x_data3, smoothedy_data3, "--");// 滤波数据绘图
    //plt::named_plot("LM Data", x_data3, lmy_data3, "r");// LM数据绘图
    //plt::title("Data 3 Process");
    //plt::legend();
    //plt::show();

    //plt::named_plot("Origin Data", x_data4, y_data4, ":");// 原始数据绘图
    //plt::named_plot("Smooth Data", x_data4, smoothedy_data4, "--");// 滤波数据绘图
    //plt::named_plot("LM Data", x_data4, lmy_data4, "r");// LM数据绘图
    //plt::title("Data 4 Process");
    //plt::legend();
    //plt::show();
 
    fftw_complex* in1, * out1;
    fftw_complex* in2, * out2;
    fftw_complex* in3, * out3;
    fftw_complex* in4, * out4;
    fftw_plan p1;
    fftw_plan p2;
    fftw_plan p3;
    fftw_plan p4;
    in1 = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * n);
    out1 = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * n);
    p1 = fftw_plan_dft_1d(n, in1, out1, FFTW_FORWARD, FFTW_MEASURE);

    in2 = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * n);
    out2 = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * n);
    p2 = fftw_plan_dft_1d(n, in2, out2, FFTW_FORWARD, FFTW_MEASURE);

    in3 = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * n);
    out3 = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * n);
    p3 = fftw_plan_dft_1d(n, in3, out3, FFTW_FORWARD, FFTW_MEASURE);

    in4 = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * n);
    out4 = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * n);
    p4 = fftw_plan_dft_1d(n, in4, out4, FFTW_FORWARD, FFTW_MEASURE);

   
    for (i = 0; i < n; i++)
    {
        in1[i][0] = lmy_data1[i];
        in1[i][1] = 1;

        in2[i][0] = lmy_data2[i];
        in2[i][1] = 1;

        in3[i][0] = lmy_data3[i];
        in3[i][1] = 1;

        in4[i][0] = lmy_data4[i];
        in4[i][1] = 1;
    }

    fftw_execute(p1);
    fftw_execute(p2);
    fftw_execute(p3);
    fftw_execute(p4);
    fftw_destroy_plan(p1);
    fftw_destroy_plan(p2);
    fftw_destroy_plan(p3);
    fftw_destroy_plan(p4);



    //// 将 lmy_data1 复制到 GPU 内存
    //double* d_lmy_data1;
    //hipMalloc((void**)&d_lmy_data1, Nl * sizeof(double));
    //hipMemcpy(d_lmy_data1, lmy_data1.data(), Nl * sizeof(double), hipMemcpyHostToDevice);
    //// 创建 CUFFT 计划
    //hipfftHandle plan;
    //hipfftPlan1d(&plan, Nl, HIPFFT_Z2Z, 1);
    //// 执行 FFT
    //hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_lmy_data1, (hipfftDoubleComplex*)d_lmy_data1, HIPFFT_FORWARD);
    //// 将结果传回主机
    //hipMemcpy(lmy_data1.data(), d_lmy_data1, Nl * sizeof(double), hipMemcpyDeviceToHost);

    //writeCSV(lmy_data1, "lmy_data1_fft.csv");

    //// 将 lmy_data2 复制到 GPU 内存
    //double* d_lmy_data2;
    //hipMalloc((void**)&d_lmy_data2, Nl * sizeof(double));
    //hipMemcpy(d_lmy_data2, lmy_data2.data(), Nl * sizeof(double), hipMemcpyHostToDevice);
    //// 创建 CUFFT 计划
    //
    ////hipfftPlan1d(&plan1, Nl, HIPFFT_Z2Z, 1);
    //// 执行 FFT
    //hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_lmy_data2, (hipfftDoubleComplex*)d_lmy_data2, HIPFFT_FORWARD);
    //// 将结果传回主机
    //hipMemcpy(lmy_data2.data(), d_lmy_data2, Nl * sizeof(double), hipMemcpyDeviceToHost);
    //
    //// 将 lmy_data3 复制到 GPU 内存
    //double* d_lmy_data3;
    //hipMalloc((void**)&d_lmy_data3, Nl * sizeof(double));
    //hipMemcpy(d_lmy_data3, lmy_data3.data(), Nl * sizeof(double), hipMemcpyHostToDevice);
    //// 创建 CUFFT 计划
    //
    ////hipfftPlan1d(&plan1, Nl, HIPFFT_Z2Z, 1);
    //// 执行 FFT
    //hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_lmy_data3, (hipfftDoubleComplex*)d_lmy_data3, HIPFFT_FORWARD);
    //// 将结果传回主机
    //hipMemcpy(lmy_data3.data(), d_lmy_data3, Nl * sizeof(double), hipMemcpyDeviceToHost);
    //
    //// 将 lmy_data4 复制到 GPU 内存
    //double* d_lmy_data4;
    //hipMalloc((void**)&d_lmy_data4, Nl * sizeof(double));
    //hipMemcpy(d_lmy_data4, lmy_data4.data(), Nl * sizeof(double), hipMemcpyHostToDevice);
    //// 创建 CUFFT 计划
    //
    ////hipfftPlan1d(&plan1, Nl, HIPFFT_Z2Z, 1);
    //// 执行 FFT
    //hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_lmy_data4, (hipfftDoubleComplex*)d_lmy_data4, HIPFFT_FORWARD);
    //// 将结果传回主机
    //hipMemcpy(lmy_data4.data(), d_lmy_data4, Nl * sizeof(double), hipMemcpyDeviceToHost);

    //// 销毁 CUFFT 计划和 GPU 内存
    //hipfftDestroy(plan);
    //
    //hipFree(d_lmy_data1);
    //hipFree(d_lmy_data2);
    //hipFree(d_lmy_data3);
    //hipFree(d_lmy_data4);

    //double fs = 1 / 0.0001220703125; //采样率
    double fs = 1000; //采样率

    
    // 创建频率轴1
    int L = n/2;
    std::vector<double> x1(L), y1(L);
    std::vector<double> x2(L), y2(L);
    std::vector<double> x3(L), y3(L);
    std::vector<double> x4(L), y4(L);
    
    for (int i = 0; i < L; ++i){

        x1.at(i) = fs * i / n;
        y1.at(i) = abs(out1[i][0]) * 2.0 / n;

    }
    // 使用max_element 查找最大值的迭代器
    auto maxElement1 = max_element(y1.begin(), y1.end());
   
    //cout << "最大值是: " << *maxElement1 << std::endl;

    // 创建频率轴2
    for (int i = 0; i < L; ++i) {

        x2.at(i) = fs * i / n;
        y2.at(i) = abs(out1[i][0]) * 2.0 / n;

    }
    // 使用max_element 查找最大值的迭代器
    auto maxElement2 = max_element(y2.begin(), y2.end());

    //cout << "最大值是: " << *maxElement2 << std::endl;

    // 创建频率轴3
    for (int i = 0; i < L; ++i) {

        x3.at(i) = fs * i / n;
        y3.at(i) = abs(out1[i][0]) * 2.0 / n;

    }
    // 使用max_element 查找最大值的迭代器
    auto maxElement3 = max_element(y3.begin(), y3.end());

    //cout << "最大值是: " << *maxElement3 << std::endl;

    // 创建频率轴4
    for (int i = 0; i < L; ++i) {

        x4.at(i) = fs * i / n;
        y4.at(i) = abs(out1[i][0]) * 2.0 / n;

    }
    // 使用max_element 查找最大值的迭代器
    auto maxElement4 = max_element(y4.begin(), y4.end());

    //cout << "最大值是: " << *maxElement4 << std::endl;
    

    // 滑动平均滤波
   // int windowSize2 = 10; // 设置滑动窗口大小
   //vector<double> smoothedY1 = movingAverage(y1, windowSize2);

   //vector<double> smoothedY2 = movingAverage(y2, windowSize2);

   //vector<double> smoothedY3 = movingAverage(y3, windowSize2);

   //vector<double> smoothedY4 = movingAverage(y4, windowSize2);


    
    double y = *maxElement1;
    int maxIndex = distance(y1.begin(), maxElement1);
    //double x_d = x1[maxIndex];
    printf("%lf,%lf", x1[maxIndex], y);

    // 使用matplotlibcpp绘制结果   
    plt::plot(x1, y1); // 使用滤波后的数据进行绘图
    //plt::xlim(-100, 3000);
    plt::xlabel("Frequency/hz");
    plt::ylabel("Amplitude");
   // plt::grid('b');
    plt::title("Data 1 Frequency-Amplitude");
    plt::text(10, 1, "(8.333333,1.008935)");
    plt::show();

    plt::plot(x2, y2);
    plt::xlabel("Frequency/hz");
    plt::ylabel("Amplitude");
    plt::title("Data 2 Frequency-Amplitude");
    //plt::show();

    plt::plot(x3, y3);
    plt::xlabel("Frequency/hz");
    plt::ylabel("Amplitude");
    plt::title("Data 3 Frequency-Amplitude");
    //plt::show();

    plt::plot(x4, y4);
    plt::xlabel("Frequency/hz");
    plt::ylabel("Amplitude");
    plt::title("Data 4 Frequency-Amplitude");
    //plt::show();

    Eigen::Matrix2d A(2, 2);
    Eigen::Vector2d b(2, 1);

    A(0, 0) = *maxElement1;
    A(0, 1) = *maxElement2;
    A(1, 0) = *maxElement3;
    A(1, 1) = *maxElement4;

    b(0, 0) = *maxElement3 + *maxElement1;
    b(1, 0) = *maxElement4 + *maxElement2;
    cout << "Matrix A:\n" << A << endl;
    cout << "Vector b:\n" << b << endl;

    double a, D;
    double Omega_n = 60;//hz
    double Delta_t = 0.001;//s
    double Ne;

    Eigen::Vector2d x = A.colPivHouseholderQr().solve(b);
    a = x(0, 0);
    D = x(1, 0) / (2 + a);
    Ne = pow(cos(-a / 2), -1) / (Omega_n * Delta_t);

    cout << "The a is:\n" << a << endl;
    cout << "The D is:\n" << D << endl;
    cout << "The Ne is:\n" << Ne << endl;

    return 0;
}